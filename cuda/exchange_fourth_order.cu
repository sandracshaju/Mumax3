#include "hip/hip_runtime.h"
#include <stdint.h>
#include "exchange.h"
#include "float3.h"
#include "stencil.h"
#include "amul.h"

// See exchange_fourth_order.go for more details.

extern "C" __global__ void
addexchangefourthorder(float* __restrict__ Bx, float* __restrict__ By, float* __restrict__ Bz,
            float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
            float* __restrict__ Ms_, float Ms_mul,
            float* __restrict__ A, float* __restrict__ Br, float* __restrict__ C,
            uint8_t* __restrict__ regions,
            float cx, float cy, float cz, int Nx, int Ny, int Nz, uint8_t PBC) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }

    // central cell
    int    I  = idx(ix, iy, iz);
    float3 m0 = make_float3(mx[I], my[I], mz[I]);

    if (is0(m0)) {
        return;
    }

    uint8_t r0 = regions[I];
    float3  B  = make_float3(0.0,0.0,0.0);

    int i_;    // neighbor index
    float3 m_; // neighbor mag
    float A__;  // second-order exchange stiffness
    float Br__;  // fourth-order exchange stiffness
    float C__;  // fourth-order exchange stiffness
  


    //////////////////
    // Central Spin //
    //////////////////
    i_    = idx(ix, iy, iz);
    m_    = make_float3(mx[i_], my[i_], mz[i_]);                                    // load m
    m_    = ( is0(m_)? m0: m_ );                                                    // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   =5.2 e -32;
    B    -= 4 * A__ * (1/(cx*cx) + 1/(cy*cy) + 1/(cz*cz)) * m_;
    B    -= 12 * Br__ * (1/(cx*cx*cx*cx) + 1/(cy*cy*cy*cy) + 1/(cz*cz*cz*cz)) * m_;
    B    -= 8 * C__ * (1/(cx*cx*cy*cy) + 1/(cx*cx*cz*cz) + 1/(cy*cy*cz*cz)) * m_;
    B    += 16 * Br__ * (1/(cx*cx*cy*cy) + 1/(cx*cx*cz*cz) + 1/(cy*cy*cz*cz)) * m_;
    


    ///////////////////////////////
    // Direct Nearest Neighbours //
    ///////////////////////////////
    
    // Left neighbour
    i_    = idx(lclampx(ix-1), iy, iz);                                             // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);                                    // load m
    m_    = ( is0(m_)? m0: m_ );                                                    // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= 8 * C__ * (1/(cx*cx*cy*cy) + 1/(cx*cx*cz*cz)) *m_;;
    B    += 2 * A__ / (cx*cx)) * m_ + 8 * Br__ / (cx*cx*cx*cx) * m_ + 4 * C__ * (1/(cx*cx*cy*cy) + 1/(cx*cx*cz*cz)) * m_;

    // Right neighbour
    i_    = idx(hclampx(ix+1), iy, iz);                                             // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);                                    // load m
    m_    = ( is0(m_)? m0: m_ );                                                    // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= 8 * C__ * (1/(cx*cx*cy*cy) + 1/(cx*cx*cz*cz)) *m_;;
    B    += 2 * A__ / (cx*cx)) * m_ + 8 * Br__ / (cx*cx*cx*cx) * m_ + 4 * C__ * (1/(cx*cx*cy*cy) + 1/(cx*cx*cz*cz)) * m_;

    // Below neighbour
    i_    = idx(ix, lclampy(iy-1), iz);                                             // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);                                    // load m
    m_    = ( is0(m_)? m0: m_ );                                                    // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= 8 * C__ * (1/(cy*cy*cx*cx) + 1/(cy*cy*cz*cz)) *m_;;
    B    += 2 * A__ / (cy*cy)) * m_ + 8 * Br__ / (cy*cy*cy*cy) * m_ + 4 * C__ * (1/(cy*cy*cx*cx) + 1/(cy*cy*cz*cz)) * m_;

    // Above neighbour
    i_    = idx(ix, hclampy(iy+1), iz);                                             // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);                                    // load m
    m_    = ( is0(m_)? m0: m_ );                                                    // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= 8 * C__ * (1/(cy*cy*cx*cx) + 1/(cy*cy*cz*cz)) *m_;;
    B    += 2 * A__ / (cy*cy)) * m_ + 8 * Br__ / (cy*cy*cy*cy) * m_ + 4 * C__ * (1/(cy*cy*cx*cx) + 1/(cy*cy*cz*cz)) * m_;

    // Bottom neighbour
    i_    = idx(ix, iy, lclampz(iz-1));                                             // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);                                    // load m
    m_    = ( is0(m_)? m0: m_ );                                                    // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= 8 * C__ * (1/(cz*cz*cx*cx) + 1/(cz*cz*cy*cy)) *m_;;
    B    += 2 * A__ / (cz*cz)) * m_ + 8 * Br__ / (cz*cz*cz*cz) * m_ + 4 * C__ * (1/(cz*cz*cx*cx) + 1/(cz*cz*cy*cy)) * m_;_;

    // Top neighbour
    i_    = idx(ix, iy, hclampz(iz+1));                                             // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);                                    // load m
    m_    = ( is0(m_)? m0: m_ );                                                    // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= 8 * C__ * (1/(cz*cz*cx*cx) + 1/(cz*cz*cy*cy)) *m_;;
    B    += 2 * A__ / (cz*cz)) * m_ + 8 * Br__ / (cz*cz*cz*cz) * m_ + 4 * C__ * (1/(cz*cz*cx*cx) + 1/(cz*cz*cy*cy)) * m_;_;


    //////////////////////////////////////////////
    // Diagonal Nearest Neighbours in z=0 Plane //
    //////////////////////////////////////////////

    // Bottom-left neighbour
    i_    = idx(lclampx(ix-1), lclampy(iy-1), iz);  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= (2 * C__ * (1 / (cx*cx*cy*cy)) * m_;
    B    += (4 * Br__ * (1 / (cx*cx*cy*cy)) * m_;
    

    // Top-left neighbour
    i_    = idx(lclampx(ix-1), hclampy(iy+1), iz);  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= (2 * C__ * (1 / (cx*cx*cy*cy)) * m_;
    B    += (4 * Br__ * (1 / (cx*cx*cy*cy)) * m_;

    // Bottom-right neighbour
    i_    = idx(hclampx(ix+1), lclampy(iy-1), iz);  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= (2 * C__ * (1 / (cx*cx*cy*cy)) * m_;
    B    += (4 * Br__ * (1 / (cx*cx*cy*cy)) * m_;

    // Top-right neighbour
    i_    = idx(hclampx(ix+1), hclampy(iy+1), iz);  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= (2 * C__ * (1 / (cx*cx*cy*cy)) * m_;
    B    += (4 * Br__ * (1 / (cx*cx*cy*cy)) * m_;


    //////////////////////////////////////////////
    // Diagonal Nearest Neighbours in x=0 Plane //
    //////////////////////////////////////////////

    // Bottom-left neighbour
    i_    = idx(ix, lclampy(iy-1), hclampz(iz+1));  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= (2 * C__ * (1 / (cy*cy*cz*cz)) * m_;
    B    += (4 * Br__ * (1 / (cy*cy*cz*cz)) * m_;

    // Top-left neighbour
    i_    = idx(ix, hclampy(iy+1), hclampz(iz+1));  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= (2 * C__ * (1 / (cy*cy*cz*cz)) * m_;
    B    += (4 * Br__ * (1 / (cy*cy*cz*cz)) * m_;

    // Bottom-right neighbour
    i_    = idx(ix, lclampy(iy-1), lclampz(iz-1));  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= (2 * C__ * (1 / (cy*cy*cz*cz)) * m_;
    B    += (4 * Br__ * (1 / (cy*cy*cz*cz)) * m_;

    // Top-right neighbour
    i_    = idx(ix, hclampy(iy+1), lclampz(iz-1));  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= (2 * C__ * (1 / (cy*cy*cz*cz)) * m_;
    B    += (4 * Br__ * (1 / (cy*cy*cz*cz)) * m_;


    //////////////////////////////////////////////
    // Diagonal Nearest Neighbours in y=0 Plane //
    //////////////////////////////////////////////

    // Bottom-left neighbour
    i_    = idx(lclampx(ix-1), iy, hclampz(iz+1));  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= (2 * C__ * (1 / (cx*cx*cz*cz)) * m_;
    B    += (4 * Br__ * (1 / (cx*cx*cz*cz)) * m_;
    // Top-left neighbour
    i_    = idx(lclampx(ix-1), iy, lclampz(iz-1));  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= (2 * C__ * (1 / (cx*cx*cz*cz)) * m_;
    B    += (4 * Br__ * (1 / (cx*cx*cz*cz)) * m_;

    // Bottom-right neighbour
    i_    = idx(hclampx(ix+1), iy, hclampz(iz+1));  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= (2 * C__ * (1 / (cx*cx*cz*cz)) * m_;
    B    += (4 * Br__ * (1 / (cx*cx*cz*cz)) * m_;
    // Top-right neighbour
    i_    = idx(hclampx(ix+1), iy, lclampz(iz-1));  // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);    // load m
    m_    = ( is0(m_)? m0: m_ );                    // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= (2 * C__ * (1 / (cx*cx*cz*cz)) * m_;
    B    += (4 * Br__ * (1 / (cx*cx*cz*cz)) * m_;


    ///////////////////////////////////////
    // Next-Next-Next Nearest Neighbours //
    ///////////////////////////////////////

    // Two over to left
    i_    = idx(lclampx(ix-2), iy, iz);           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= (2 * Br__ / (cx*cx*cx*cx)) * m_;

    // Two over to right
    i_    = idx(hclampx(ix+2), iy, iz);           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= (2 * Br__ / (cx*cx*cx*cx)) * m_;

    // Two below
    i_    = idx(ix, lclampy(iy-2), iz);           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= (2 * Br__ / (cy*cy*cy*cy)) * m_;

    // Two above
    i_    = idx(ix, hclampy(iy+2), iz);           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= (2 * Br__ / (cy*cy*cy*cy)) * m_;

    // Two bottom
    i_    = idx(ix, iy, lclampz(iz-2));           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= (2 * Br__ / (cz*cz*cz*cz)) * m_;

    // Two top
    i_    = idx(ix, iy, hclampz(iz+2));           // clamps or wraps index according to PBC
    m_    = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_    = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    A__   = 1.5 e -14;
    Br__  = 0.8 e -32;
    C__   = 5.2 e -32;
    B    -= (2 * Br__ / (cz*cz*cz*cz)) * m_;


    float invMs = inv_Msat(Ms_, Ms_mul, I);

    Bx[I] += B.x*invMs;
    By[I] += B.y*invMs;
    Bz[I] += B.z*invMs;

}
