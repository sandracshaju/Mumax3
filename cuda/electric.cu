#include "hip/hip_runtime.h"
#include <stdint.h>
#include "exchange.h"
#include "stencil.h"
#include "amul.h"

// Electric field term according to
// Katsura, Nagaosa, Balatsky, Phys. Rev. Lett. 95, 057205 (2005).

extern "C" __global__ void
addelectric(float* __restrict__ Bx, float* __restrict__ By, float* __restrict__ Bz,
            float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
            float* __restrict__ Ms_, float Ms_mul,
            float* __restrict__ eLUT2d, uint8_t* __restrict__ regions,
            float cx, float cy, float cz, int Nx, int Ny, int Nz, uint8_t PBC) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }

    // central cell
    int I = idx(ix, iy, iz);
    float3 m0 = make_float3(mx[I], my[I], mz[I]);

    if (is0(m0)) {
        return;
    }

    uint8_t r0 = regions[I];
    float3 B  = make_float3(0.0,0.0,0.0);

    int i_;    // neighbor index
    float3 m_; // neighbor mag
    float ered__; // reduced electric field


    ///////////////////
    // x derivatives //
    ///////////////////

    // right neighbor
    i_  = idx(hclampx(ix+1), iy, iz);           // clamps or wraps index according to PBC
    m_  = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_  = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    ered__ = eLUT2d[symidx(r0, regions[i_])];
    B.x -= (ered__/cx) * m_.z;
    B.z += (ered__/cx) * m_.x;

    // left neighbor
    i_  = idx(lclampx(ix-1), iy, iz);           // clamps or wraps index according to PBC
    m_  = make_float3(mx[i_], my[i_], mz[i_]);  // load m
    m_  = ( is0(m_)? m0: m_ );                  // replace missing non-boundary neighbor
    ered__ = eLUT2d[symidx(r0, regions[i_])];
    B.x += (ered__/cx) * m_.z;
    B.z -= (ered__/cx) * m_.x;
    
    ///////////////////
    // y derivatives //
    ///////////////////

    // above neighbor
    i_  = idx(ix, hclampy(iy+1), iz);
    m_  = make_float3(mx[i_], my[i_], mz[i_]);
    m_  = ( is0(m_)? m0: m_ );
    ered__ = eLUT2d[symidx(r0, regions[i_])];
    B.y -= (ered__/cy) * m_.z;
    B.z += (ered__/cy) * m_.y;

    // below neighbor
    i_  = idx(ix, lclampy(iy-1), iz);
    m_  = make_float3(mx[i_], my[i_], mz[i_]);
    m_  = ( is0(m_)? m0: m_ );
    ered__ = eLUT2d[symidx(r0, regions[i_])];
    B.y += (ered__/cy) * m_.z;
    B.z -= (ered__/cy) * m_.y;


    float invMs = inv_Msat(Ms_, Ms_mul, I);
    Bx[I] += B.x*invMs;
    By[I] += B.y*invMs;
    Bz[I] += B.z*invMs;
}
